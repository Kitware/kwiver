#include "hip/hip_runtime.h"
/*ckwg +29
* Copyright 2016 by Kitware SAS, 2018-2019 Kitware, Inc.
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*
*  * Redistributions of source code must retain the above copyright notice,
*    this list of conditions and the following disclaimer.
*
*  * Redistributions in binary form must reproduce the above copyright notice,
*    this list of conditions and the following disclaimer in the documentation
*    and/or other materials provided with the distribution.
*
*  * Neither name of Kitware, Inc. nor the names of any contributors may be used
*    to endorse or promote products derived from this software without specific
*    prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS ``AS IS''
* AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
* ARE DISCLAIMED. IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE LIABLE FOR
* ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
* DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
* SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
* CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
* OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef INTEGRATE_DEPTH_MAPS_CU_
#define INTEGRATE_DEPTH_MAPS_CU_

// STD include
#include <math.h>
#include <stdio.h>
#include <vector>
#include "cuda_error_check.h"

#define size4x4 16

//*****************************************************************************

// Define texture and constants
__constant__ double c_gridOrig[3];        // Origin of the output volume
__constant__ int3 c_gridDims;             // Dimensions of the output volume
__constant__ double c_gridSpacing[3];     // Spacing of the output volume
__constant__ int2 c_depthMapDims;         // Dimensions of all depths map
__constant__ double c_rayPotentialThick;  // Thickness threshold for the ray potential function
__constant__ double c_rayPotentialRho;    // Rho at the Y axis for the ray potential function
__constant__ double c_rayPotentialEta;
__constant__ double c_rayPotentialEpsilon;
__constant__ double c_rayPotentialDelta;
int grid_dims[3];


//*****************************************************************************
//     Truncated Signed Distance Function (TSDF) Parameter Description
//*****************************************************************************
//** Eta is a percentage of rho ( 0 < Eta < 1)
//** Epsilon is a percentage of rho ( 0 < Epsilon < 1)
//** Delta has to be superior to Thick
//
//                     'real distance' - 'depth value'
//                                     |
//                                     |
//                                     |         ---------------  Rho
//                                     |        /|             |
//                                     |       /               |
//                                     |      /  |             |
//                                     |     /                 |
//                                     |    /    |             |
//                                     |   /                   |
//                                     |  /      |             |
//                                     | /         Epsilon*Rho |______________
//                                     |/        |
//----------------------------------------------------------------------------
//                                    /
//                                   /
//                                  /
//--------------  Eta*rho          /
//             |                  /
//             |                 /
//             |                /
//             |               /
//             |              /
//             ---------------
//                            <--------->
//                               Thick
//             <----------------------->
//                        Delta
//*****************************************************************************

__device__ void computeVoxelCenter(int voxelCoordinate[3], double output[3])
{
  output[0] = c_gridOrig[0] + (voxelCoordinate[0] + 0.5) * c_gridSpacing[0];
  output[1] = c_gridOrig[1] + (voxelCoordinate[1] + 0.5) * c_gridSpacing[1];
  output[2] = c_gridOrig[2] + (voxelCoordinate[2] + 0.5) * c_gridSpacing[2];
}

//*****************************************************************************

//Apply a 3x4 matrix to a 3D points (assumes last row of M is 0, 0, 0, 1)
__device__ void transformFrom4Matrix(double M[size4x4], double point[3], double output[3])
{
  output[0] = M[0 * 4 + 0] * point[0] + M[0 * 4 + 1] * point[1] + M[0 * 4 + 2] * point[2] + M[0 * 4 + 3];
  output[1] = M[1 * 4 + 0] * point[0] + M[1 * 4 + 1] * point[1] + M[1 * 4 + 2] * point[2] + M[1 * 4 + 3];
  output[2] = M[2 * 4 + 0] * point[0] + M[2 * 4 + 1] * point[1] + M[2 * 4 + 2] * point[2] + M[2 * 4 + 3];
}

//*****************************************************************************

// Compute the norm of a 3 vec
__device__ double norm(double vec[3])
{
  return sqrt(vec[0] * vec[0] + vec[1] * vec[1] + vec[2] * vec[2]);
}

//*****************************************************************************

//Ray potential function which computes the increment to the current voxel
__device__ void rayPotential(double realDistance, double depthMapDistance, double& res)
{
  double diff = (realDistance - depthMapDistance);

  double absoluteDiff = abs(diff);
  // Can't divide by zero
  int sign = diff != 0 ? diff / absoluteDiff : 0;

  if (absoluteDiff > c_rayPotentialDelta)
    res = diff > 0 ? c_rayPotentialEpsilon * c_rayPotentialRho
                   : - c_rayPotentialEta * c_rayPotentialRho;
  else if (absoluteDiff > c_rayPotentialThick)
    res = c_rayPotentialRho * sign;
  else
    res = (c_rayPotentialRho / c_rayPotentialThick) * diff;
}

//*****************************************************************************

// Compute the voxel Id on a 1D table according to its 3D coordinates
__device__ int computeVoxelIDGrid(int coordinates[3])
{
  int dimX = c_gridDims.x;
  int dimY = c_gridDims.y;
  int i = coordinates[0];
  int j = coordinates[1];
  int k = coordinates[2];
  return (k*dimY + j)*dimX + i;
}

//*****************************************************************************

//Compute the pixel Id on a 1D table according to its 3D coordinates (third coordinate is not used)
__device__ int computeVoxelIDDepth(int coordinates[3])
{
  int dimX = c_depthMapDims.x;
  int dimY = c_depthMapDims.y;
  int x = coordinates[0];
  int y = coordinates[1];
  // /!\ vtkImageData has its origin at the bottom left, not top left
  return (dimX*(dimY - 1 - y)) + x;
}

//*****************************************************************************

// Main kernel for adding a depth map to the volume
__global__ void depthMapKernel(double* depths, double* weights, double matrixK[size4x4], double matrixRT[size4x4],
  double* output)
{
  // Get voxel coordinate according to thread id
  int voxelIndex[3] = { (int)threadIdx.x, (int)blockIdx.y, (int)blockIdx.z };

  double voxelCenterCoordinate[3];
  computeVoxelCenter(voxelIndex, voxelCenterCoordinate);

  // Transform voxel center from real coord to camera coords
  double voxelCenterCamera[3];
  transformFrom4Matrix(matrixRT, voxelCenterCoordinate, voxelCenterCamera);

  // Transform voxel center from camera coords to depth map homogeneous coords
  double voxelCenterHomogen[3];
  transformFrom4Matrix(matrixK, voxelCenterCamera, voxelCenterHomogen);
  if (voxelCenterHomogen[2] < 0)
    return;

  // Get voxel center on depth map coord
  double voxelCenterDepthMap[2];
  voxelCenterDepthMap[0] = voxelCenterHomogen[0] / voxelCenterHomogen[2];
  voxelCenterDepthMap[1] = voxelCenterHomogen[1] / voxelCenterHomogen[2];
  // Get real pixel position (approximation)
  int pixel[3];
  pixel[0] = round(voxelCenterDepthMap[0]);
  pixel[1] = round(voxelCenterDepthMap[1]);
  pixel[2] = 0;

  // Test if coordinate are inside depth map
  if (pixel[0] < 0 || pixel[1] < 0 || pixel[0] >= c_depthMapDims.x || pixel[1] >= c_depthMapDims.y)
    return;

  // Compute the ID on depthmap values according to pixel position and depth map dimensions
  int depthMapId = computeVoxelIDDepth(pixel);
  double depth = depths[depthMapId];
  double weight = weights ? weights[depthMapId] : 1.0;
  if (depth <= 0 || weight <= 0)
    return;

  int gridId = computeVoxelIDGrid(voxelIndex);  // Get the distance between voxel and camera
  double realDepth = voxelCenterCamera[2];
  double newValue;
  rayPotential(realDepth, depth, newValue);
  // Update the value to the output
  output[gridId] += weight * newValue;
}

//*****************************************************************************

// Initialize cuda constants
void cuda_initalize(int h_gridDims[3],     // Dimensions of the output volume
          double h_gridOrig[3],  // Origin of the output volume
          double h_gridSpacing[3], // Spacing of the output volume
          double h_rayPThick,
          double h_rayPRho,
          double h_rayPEta,
          double h_rayPEpsilon,
          double h_rayPDelta)
{
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_gridDims), h_gridDims, 3 * sizeof(int)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_gridOrig), h_gridOrig, 3 * sizeof(double)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_gridSpacing), h_gridSpacing, 3 * sizeof(double)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialThick), &h_rayPThick, sizeof(double)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialRho), &h_rayPRho, sizeof(double)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialEta), &h_rayPEta, sizeof(double)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialEpsilon), &h_rayPEpsilon, sizeof(double)));
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialDelta), &h_rayPDelta, sizeof(double)));

  grid_dims[0] = h_gridDims[0];
  grid_dims[1] = h_gridDims[1];
  grid_dims[2] = h_gridDims[2];
}

//*****************************************************************************

void launch_depth_kernel(double * d_depth, double * d_conf, int h_depthMapDims[2], double d_K[size4x4], double d_RT[size4x4], double* d_volume)
{
  // Organize threads into blocks and grids
  dim3 dimBlock(grid_dims[0], 1, 1); // nb threads on each block
  dim3 dimGrid(1, grid_dims[1], grid_dims[2]); // nb blocks on a grid
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_depthMapDims), h_depthMapDims, 2 * sizeof(int)));
  CudaErrorCheck(hipDeviceSynchronize());
  depthMapKernel << < dimGrid, dimBlock >> >(d_depth, d_conf, d_K, d_RT, d_volume);
  CudaErrorCheck(hipPeekAtLastError());
  CudaErrorCheck(hipDeviceSynchronize());
}


#endif
